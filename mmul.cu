#include "hip/hip_runtime.h"
/*************************************************************************
/* ECE 277: GPU Programmming 2020 
/* Author and Instructer: Cheolhong An
/* Copyright 2020
/* University of California, San Diego
/*************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernal_mmul(int* A, int* W, int* C, const int M, const int N, const int K);

void cu_mmul(int* A, int* W, int* C, const int M, const int N, const int K)
{
	int *d_a, *d_w, *d_c;

	dim3 blk;
	blk.x = 32; blk.y = 32;

    //const int M = 34*34, N = 64, K = 64;
	dim3 grid;
	grid.x = (M + blk.x - 1) / blk.x;
	grid.y = (N + blk.y - 1) / blk.y;
	grid.z = 1;

	int A_size = sizeof(unsigned int)*M*K;
	int W_size = sizeof(unsigned int)*K*N;
	int C_size = sizeof(unsigned int)*M*N;

	hipMalloc((void **)&d_a, A_size);
	hipMalloc((void **)&d_w, W_size);
	hipMalloc((void **)&d_c, C_size);

	hipMemcpy(d_a, A, A_size, hipMemcpyHostToDevice);
	hipMemcpy(d_w, W, W_size, hipMemcpyHostToDevice);

	kernal_mmul << < grid, blk >> > (d_a, d_w, d_c, M, N, K);

	hipMemcpy(C, d_c, C_size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_w);
	hipFree(d_c);
}

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
__global__ void kernal_mmul(int* A, int* W, int* C, const int M, const int N, const int K) {

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    if (m < M && N < N) {
        int psum = 0;
		#pragma unroll
        for (int k = 0; k < K; k++) {
            psum += A[OFFSET(m, k, K)] * W[OFFSET(k, n, N)];
        }
        C[OFFSET(m, n, N)] = psum;
    }
}

